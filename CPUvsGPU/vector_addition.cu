#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <omp.h>

#define N 100000000

void add_cpu(float *x, float *y, float *z) {
    #pragma omp parallel for
    for (int i = 0; i < N; i++) {
        z[i] = x[i] + y[i];
    }
}

__global__ void add_gpu(float *x, float *y, float *z) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        z[i] = x[i] + y[i];
    }
}

int main() {
    float *x, *y, *z_cpu, *z_gpu;
    size_t size = N * sizeof(float);

    // Allocate memory for arrays on host (CPU)
    x = (float *) malloc(size);
    y = (float *) malloc(size);
    z_cpu = (float *) malloc(size);
    z_gpu = (float *) malloc(size);

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Measure time taken by CPU with OpenMP
    LARGE_INTEGER start, end, frequency;
    QueryPerformanceFrequency(&frequency);
    QueryPerformanceCounter(&start);
    add_cpu(x, y, z_cpu);
    QueryPerformanceCounter(&end);
    double cpu_time_omp = (end.QuadPart - start.QuadPart) / (double) frequency.QuadPart;
    printf("Time taken by CPU with OpenMP: %.6f seconds\n", cpu_time_omp);

    // Allocate memory for arrays on device (GPU)
    float *d_x, *d_y, *d_z;
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);

    // Copy input data from host to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Measure time taken by GPU
    QueryPerformanceCounter(&start);
    add_gpu<<<(N + 255) / 256, 256>>>(d_x, d_y, d_z);
    hipDeviceSynchronize();
    QueryPerformanceCounter(&end);
    double gpu_time = (end.QuadPart - start.QuadPart) / (double) frequency.QuadPart;
    printf("Time taken by GPU: %.6f seconds\n", gpu_time);

    // Copy output data from device to host
    hipMemcpy(z_gpu, d_z, size, hipMemcpyDeviceToHost);

    // Verify correctness of GPU results
    for (int i = 0; i < N; i++) {
        if (z_cpu[i] != z_gpu[i]) {
            printf("Error: GPU and CPU results do not match\n");
            break;
        }
    }

    // Free memory on host and device
    free(x);
    free(y);
    free(z_cpu);
    free(z_gpu);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}
